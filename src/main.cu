#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <stdint.h>

#include <X11/Xlib.h>

#include <GL/gl.h>
#include <GL/glx.h>

#define IMGUI_DISABLE_SSE
#include "imgui.h"
#include "imgui_impl_xlib.h"
#include "imgui_impl_opengl2.h"

#include "imgui.cpp"
#include "imgui_draw.cpp"
#include "imgui_tables.cpp"
#include "imgui_widgets.cpp"
#include "imgui_demo.cpp"
#include "imgui_impl_xlib.cpp"
#include "imgui_impl_opengl2.cpp"

#include "defer.h"
#include "state.h"
#include "time_utils.h"
#include "logging.h"

constexpr float TARGET_SECONDS_PER_FRAME = 1.0f/20.0f;
constexpr int   WINDOW_WIDTH             = 800;
constexpr int   WINDOW_HEIGHT            = 600;

template <typename T>
void swap(T &a, T &b)
{
    T tmp = a;
    a = b;
    b = tmp;
}

__global__ void
heat_conduction_kernel(
    int width, int height, float alphaTimesDt,
    float* temp_in, float* temp_out)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int col = idx % width;
    int row = idx / width;

	if (0 < row && row < height-1 && 0 < col && col < width-1) {
		int left  = idx - 1;
		int right = idx + 1;
		int up    = idx - width;
		int down  = idx + width;

		// evaluate derivatives
		float d2tdx2 = temp_in[left] - 2.0f * temp_in[idx] + temp_in[right];
		float d2tdy2 = temp_in[up]   - 2.0f * temp_in[idx] + temp_in[down];

        if (d2tdx2 < 1e-5) { d2tdx2 = 0.0f; }
        if (d2tdy2 < 1e-5) { d2tdy2 = 0.0f; }

		// update temperature
        temp_out[idx] = temp_in[idx] + alphaTimesDt * (d2tdx2 + d2tdy2);
	}

}

__global__ void
color_kernel(
    int width, int height, float lighter_temp,
    float *temps, uint32_t *pixels_out)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < width * height) {
        float norm_temp = temps[idx]/lighter_temp;

        // Pixel format: 0xAABBGGRR
        uint32_t color = 0xff000000;

        int grey = static_cast<int>(roundf(norm_temp * 255.0f)) & 0xff;
        color = grey << (0 * 8)  // Red
              | grey << (1 * 8)  // Green
              | grey << (2 * 8)  // Blue
              | 0xff << (3 * 8); // Alpha

        pixels_out[idx] = color;
    }
}

bool check_last_cuda_error()
{
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess)
    {
        loge("CUDA Runtime Error: %s", hipGetErrorString(err));
        return false;
    }
    return true;
}

int main(int argc, char *argv[])
{
    ApplicationState app_state     = {};
    app_state.alpha                = 0.5f;
    app_state.lighter_temp         = 1.0f;
    app_state.speed_multiplier     = 1;
    app_state.show_settings_window = true;

    app_state.host_pixel_buffer = static_cast<uint32_t *>(malloc(WINDOW_WIDTH*WINDOW_HEIGHT*sizeof(uint32_t)));
    memset(app_state.host_pixel_buffer, 0, WINDOW_WIDTH*WINDOW_HEIGHT*sizeof(uint32_t));
    defer(free(app_state.host_pixel_buffer));

    hipMalloc(&app_state.device_pixel_buffer, WINDOW_WIDTH*WINDOW_HEIGHT*sizeof(uint32_t));
    if (!check_last_cuda_error()) { return 1; }
    hipMemset(app_state.device_pixel_buffer, 0, WINDOW_WIDTH*WINDOW_HEIGHT*sizeof(uint32_t));
    defer(hipFree(app_state.device_pixel_buffer));

    hipMalloc(&app_state.primary_temp_buffer, WINDOW_WIDTH*WINDOW_HEIGHT*sizeof(float));
    if (!check_last_cuda_error()) { return 1; }
    hipMemset(app_state.primary_temp_buffer, 0, WINDOW_WIDTH*WINDOW_HEIGHT*sizeof(float));
    defer(hipFree(app_state.primary_temp_buffer));

    hipMalloc(&app_state.secondary_temp_buffer, WINDOW_WIDTH*WINDOW_HEIGHT*sizeof(float));
    if (!check_last_cuda_error()) { return 1; }
    hipMemset(app_state.secondary_temp_buffer, 0, WINDOW_WIDTH*WINDOW_HEIGHT*sizeof(float));
    defer(hipFree(app_state.secondary_temp_buffer));
    hipDeviceSynchronize();

    if (!app_state.host_pixel_buffer
        || !app_state.device_pixel_buffer
        || !app_state.primary_temp_buffer
        || !app_state.secondary_temp_buffer) {
        loge("Failed to allocate buffers");
        return 1;
    }

    Display *display = XOpenDisplay(0);
    if (!display) {
        loge("Failed to open display");
        return 1;
    }
    defer(XCloseDisplay(display));

    Window window = XCreateSimpleWindow(
        display, XDefaultRootWindow(display), 0, 0,
        WINDOW_WIDTH, WINDOW_HEIGHT, 0, 0, 0);
    defer(XDestroyWindow(display, window));

    Atom wm_delete_window = XInternAtom(display, "WM_DELETE_WINDOW", false);
    XSetWMProtocols(display, window, &wm_delete_window, 1);

    GLXContext gl_context = {};
    {
        int n = 0;
        XVisualInfo *visual_info = XGetVisualInfo(display, 0, 0, &n);
        gl_context = glXCreateContext(display, visual_info, NULL, GL_TRUE);
    }
    if (!gl_context) {
        loge("Failed to create OpenGL2 context");
        return 1;
    }
    defer(glXDestroyContext(display, gl_context));

    if (!glXMakeCurrent(display, window, gl_context)) {
        loge("Failed to attach the OpenGL2 context to the window");
        return 1;
    }

    logi("GL Renderer:  %s", glGetString(GL_RENDERER));
    logi("GL Version:   %s", glGetString(GL_VERSION));
    logi("GLSL Version: %s", glGetString(GL_SHADING_LANGUAGE_VERSION));

    glGenTextures(1, &app_state.texture_handle);
    glViewport(0, 0, WINDOW_WIDTH, WINDOW_HEIGHT);

    XSelectInput(display, window,
        ExposureMask|KeyPressMask|KeyReleaseMask|
        PointerMotionMask|ButtonPressMask|ButtonReleaseMask);
    XStoreName(display, window, "CUDA Heat Conduction Demo");
    XMapWindow(display, window);

    if(!IMGUI_CHECKVERSION()) {
        loge("IMGUI_CHECKVERSION failed");
        return 1;
    }

    if (!ImGui::CreateContext()) {
        loge("Failed to create ImGui context");
        return 1;
    }
    defer(ImGui::DestroyContext());

    if (!ImGui_ImplXlib_InitForOpenGL(display, window, gl_context)) {
        loge("Failed to initialize ImGui for Xlib");
        return 1;
    }
    defer(ImGui_ImplXlib_Shutdown);

    if (!ImGui_ImplOpenGL2_Init()) {
        loge("Failed to initialize ImGui for OpenGL2");
        return 1;
    }
    defer(ImGui_ImplOpenGL2_Shutdown());

    ImGui::StyleColorsDark();
    ImGuiIO &io = ImGui::GetIO();
    io.IniFilename = NULL;

    hipDeviceProp_t device_properties;
    if (hipGetDeviceProperties(&device_properties, 0) != hipSuccess) {
        loge("Failed to read device properties");
        return 1;
    }
    logi("GPU:          %s", device_properties.name);
    app_state.threads_per_block = device_properties.maxThreadsPerBlock;
    app_state.number_of_blocks = static_cast<int>(ceilf(static_cast<float>(WINDOW_HEIGHT*WINDOW_WIDTH)/app_state.threads_per_block));

    timespec begin_frame_time = get_wall_clock();
    timespec simulation_timer = get_wall_clock();

    while (!app_state.should_close) {
        while (XPending(display) > 0) {
            XEvent event = {};
            XNextEvent(display, &event);
            ImGui_ImplXlib_ProcessEvent(&event);
            switch (event.type)
            {
            case KeyPress: {
                XKeyEvent *key_event = (XKeyEvent *)&event;
                KeySym key_sym = XLookupKeysym(key_event, 0);
                if (key_sym == XK_F1) {
                    app_state.show_settings_window = !app_state.show_settings_window;
                }
            } break;

            case ClientMessage: {
                if ((Atom)event.xclient.data.l[0] == wm_delete_window) {
                    app_state.should_close = true;
                }
            } break;

            case ButtonPress:
            {
                XButtonPressedEvent *button_event = reinterpret_cast<XButtonPressedEvent *>(&event);
                if (button_event->button == Button3) {
                    float t = 1.0f;
                    XWindowAttributes window_attributes = {};
                    XGetWindowAttributes(display, window, &window_attributes);
                    int x = static_cast<int>(roundf(static_cast<float>(button_event->x)/window_attributes.width*WINDOW_WIDTH));
                    int y = static_cast<int>(roundf(static_cast<float>(button_event->y)/window_attributes.height*WINDOW_HEIGHT));
                    int idx = y * WINDOW_WIDTH + x;
                    hipMemcpy(app_state.primary_temp_buffer+idx, &t, sizeof(float), hipMemcpyHostToDevice);
                }
            } break;

            case Expose: {
                XWindowAttributes window_attributes = {};
                XGetWindowAttributes(display, window, &window_attributes);
                glViewport(0, 0, window_attributes.width, window_attributes.height);
            } break;

            default:
                break;
            }
        }

        {
            timespec now = get_wall_clock();
            float dt = get_seconds_elapsed(simulation_timer, now);
            simulation_timer = now;

            if (!app_state.simulation_paused) {
                for (int i = 0; i < app_state.speed_multiplier; ++i) {
                    heat_conduction_kernel<<<app_state.number_of_blocks, app_state.threads_per_block>>>(
                        WINDOW_WIDTH, WINDOW_HEIGHT, app_state.alpha * dt,
                        app_state.primary_temp_buffer, app_state.secondary_temp_buffer);
                    hipDeviceSynchronize();
                    swap(app_state.primary_temp_buffer, app_state.secondary_temp_buffer);
                }
            }

            color_kernel<<<app_state.number_of_blocks, app_state.threads_per_block>>>(
                WINDOW_WIDTH, WINDOW_HEIGHT, app_state.lighter_temp,
                app_state.primary_temp_buffer, app_state.device_pixel_buffer);
            hipDeviceSynchronize();

            hipMemcpy(app_state.host_pixel_buffer, app_state.device_pixel_buffer,
                       WINDOW_WIDTH*WINDOW_HEIGHT*sizeof(uint32_t), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();

            glBindTexture(GL_TEXTURE_2D, app_state.texture_handle);

            glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, WINDOW_WIDTH, WINDOW_HEIGHT, 0,
                         GL_RGBA, GL_UNSIGNED_BYTE, app_state.host_pixel_buffer);

            glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
            glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
            glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
            glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
            glTexEnvi(GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_MODULATE);

            glEnable(GL_TEXTURE_2D);

            glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
            glClear(GL_COLOR_BUFFER_BIT);

            glMatrixMode(GL_TEXTURE);
            glLoadIdentity();

            glMatrixMode(GL_MODELVIEW);
            glLoadIdentity();

            glMatrixMode(GL_PROJECTION);
            glLoadIdentity();

            glBegin(GL_TRIANGLES);
            glTexCoord2i(0, 1);
            glVertex2f(-1.0f, -1.0f);
            glTexCoord2i(0,  0);
            glVertex2f(-1.0f,  1.0f);
            glTexCoord2i(1, 0);
            glVertex2f(1.0f,  1.0f);
            glTexCoord2i(0, 1);
            glVertex2f(-1.0f, -1.0f);
            glTexCoord2i(1, 0);
            glVertex2f(1.0f,  1.0f);
            glTexCoord2i( 1, 1);
            glVertex2f(1.0f, -1.0f);
            glEnd();

            ImGui_ImplOpenGL2_NewFrame();
            ImGui_ImplXlib_NewFrame();
            ImGui::NewFrame();
            if (app_state.show_settings_window) {
                ImGui::Begin("Settings");
                if (ImGui::Button("Reset")) {
                    hipMemset(app_state.primary_temp_buffer, 0, WINDOW_WIDTH*WINDOW_HEIGHT*sizeof(float));
                }
                if (app_state.simulation_paused) {
                    if(ImGui::Button("Play")) {
                        app_state.simulation_paused = !app_state.simulation_paused;
                    }
                } else {
                    if(ImGui::Button("Pause")) {
                        app_state.simulation_paused = !app_state.simulation_paused;
                    }
                }
                ImGui::SliderInt("Speed multiplier", &app_state.speed_multiplier, 1, 10);
                ImGui::SliderFloat("Alpha", &app_state.alpha, 0.1f, 0.9f, "%.01f");

                ImGui::Text("Stats:");
                ImGui::Text("%.02f FPS", app_state.fps);
                ImGui::Text("%d Threads per block", app_state.threads_per_block);
                ImGui::Text("%d Blocks", app_state.number_of_blocks);
                ImGui::End();
            }
            ImGui::Render();
            ImGui_ImplOpenGL2_RenderDrawData(ImGui::GetDrawData());

            glXSwapBuffers(display, window);
        }

        // Enforce frame rate
        float frame_seconds_elapsed = get_seconds_elapsed(begin_frame_time, get_wall_clock());
        if (frame_seconds_elapsed < TARGET_SECONDS_PER_FRAME) {
            useconds_t sleep_us = static_cast<useconds_t>(1.0e6f * (TARGET_SECONDS_PER_FRAME - frame_seconds_elapsed));
            usleep(sleep_us);
            while (frame_seconds_elapsed < TARGET_SECONDS_PER_FRAME) {
                frame_seconds_elapsed = get_seconds_elapsed(begin_frame_time, get_wall_clock());
            }
        }
        app_state.fps = 1.0f / frame_seconds_elapsed;
        begin_frame_time = get_wall_clock();
    }

    return 0;
}
